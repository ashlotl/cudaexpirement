
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "math.h"
#include <pthread.h>
#include <unistd.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#define PORT    2084
#define MAXLINE 1024
/*
I will provide a word about tx,ty, and tz.
tx does not, in fact move the actual point (points are static), but defines how much mass will be donated.
After this donation, a density-based heat donation is calculated, but regardless of these trajectory values.
*/
__global__
void osmate(double tempRate, double massRate, double tempPush, double repulsionRate, int numPoints, double radius, double gConst, double * x, double * y, double * z, double * tx, double * ty, double * tz, double * newtx, double * newty, double * newtz, double * mass, double * temp, double * newmass, double * newtemp) {
  //The only part that really matters if I get the rest to work:
  int i=blockIdx.x + blockDim.x + threadIdx.x;
  for (int it=0;it<numPoints;it++) {//Iterate through all points
    double dist=sqrt((x[it]-x[i])*(x[it]-x[i])+(y[it]-y[i])*(y[it]-y[i])+(z[it]-z[i])*(z[it]-z[i]));//This could be pre-calculated (these points are static). Whether the calculation happens should be a matter of how many points there are -- if there are too many points we will run out of memory, but if there are not as many it is a good idea. I'm leaving it for another day.
    if (dist!=0) {
      //Congratulations, you are not looking at the same point.
      double toChange=gConst*mass[i]*mass[it]/dist;
      newtx[i]=tx[i]+(toChange+tempPush*(temp[i]-temp[it])+repulsionRate*(mass[i]-mass[it]))*(x[it]-x[i]);
      newty[i]=ty[i]+(toChange+tempPush*(temp[i]-temp[it])+repulsionRate*(mass[i]-mass[it]))*(y[it]-y[i]);
      newtz[i]=tz[i]+(toChange+tempPush*(temp[i]-temp[it])+repulsionRate*(mass[i]-mass[it]))*(z[it]-z[i]);
    }
  }
  for (int it=0;it<numPoints;it++) {
    //TODO make negative mass impossible.
    double massDisagreement=sqrt((x[it]-x[i])*(x[it]-x[i])/tx[i]+(y[it]-y[i])*(y[it]-y[i])/ty[i]+(z[it]-z[i])*(z[it]-z[i])/tz[i]);//TODO cases for rare 0 trajectory scenarios that would knacker it all.
    newmass[it]=mass[it]+massRate/massDisagreement;//Hell if I know whether this is right, but it seems it should intuitively work. Also grants some cohesion. Note we don't subtract from the subject sample because the mass comes from behind it.
    double dist=sqrt((x[it]-x[i])*(x[it]-x[i])+(y[it]-y[i])*(y[it]-y[i])+(z[it]-z[i])*(z[it]-z[i]));//There are a lot of redundant calculations here...
    double tempEx=tempRate*(temp[i]-temp[it])/dist;
    newtemp[it]=temp[it]+tempEx;
    newtemp[i]=temp[i]-tempEx;//Unless we want to go supernova. We should also have a mechanic where a mass increase "creates" heat, a decrease sucks it up, there is an external source of heat and heat can radiate away... Etc.
  }
}
//To tweak
double TEMP_RATE=0.1;
double MASS_RATE=0.4;
double TEMP_PUSH=0.03;
double REPULSION_RATE=1.0;
double SAMPLE_RADIUS=.1;
double GRAV_CONST=.0667;//Definitely not.

int ALTITUDE=5;//Radius of planet.
double LAYER_HEIGHT=.5;
int RES = 3; //How often to subdivide the base cube

//In generation, values may exceed maximums and minimums during simulation.
double leastAllowedMass=.001;
double mostAllowedMass=5;//We'll randomize between the two.

double leastAllowedTemp=.001;//Kelvin?
double mostAllowedTemp=50000;//A lot, hopefully not too much. Should check Planck heat.

double leastAllowedTrajectory=.0001;//Along axis, I'm not a madman
double mostAllowedTrajectory=2;

int main() {
  printf("Hello, you are hopefully simulating tectonics. If not, get out of here.\n");
  //Generate points...
  //The Declaration of Variable Allocation:
  int numSamples=0;
  for (double alt=LAYER_HEIGHT;alt<ALTITUDE;alt+=LAYER_HEIGHT) {
    //You may notice we project a cube into a sphere. Primarily I'm too lazy to generate something like an icosahedron, secondarily it would not have fine resolution (only subdivisions) if we used an icosahedron, tertiarily this allows us to check for seams, which are screaming alarms something is wrong, and fourth uv spheres get ridiculously fine detail only at the poles, which is a waste.
    numSamples+=(int)(pow(RES,3)-pow(RES-2,3));
  }
  printf("Allocating...\n");
  double * x = (double*)malloc(numSamples*sizeof(double));
  double * y = (double*)malloc(numSamples*sizeof(double));
  double * z = (double*)malloc(numSamples*sizeof(double));
  double * tx = (double*)malloc(numSamples*sizeof(double));
  double * ty = (double*)malloc(numSamples*sizeof(double));
  double * tz = (double*)malloc(numSamples*sizeof(double));
  double * temp = (double*)malloc(numSamples*sizeof(double));
  double * mass = (double*)malloc(numSamples*sizeof(double));
  double * newtx = (double*)malloc(numSamples*sizeof(double));
  double * newty = (double*)malloc(numSamples*sizeof(double));
  double * newtz = (double*)malloc(numSamples*sizeof(double));
  double * newtemp = (double*)malloc(numSamples*sizeof(double));
  double * newmass = (double*)malloc(numSamples*sizeof(double));
  //Here we have device copies:
  double * d_x;
  double * d_y;
  double * d_z;
  double * d_tx;
  double * d_ty;
  double * d_tz;
  double * d_temp;
  double * d_mass;
  double * d_newtx;
  double * d_newty;
  double * d_newtz;
  double * d_newtemp;
  double * d_newmass;
  hipMalloc(&d_x,numSamples*sizeof(float));
  hipMalloc(&d_y,numSamples*sizeof(float));
  hipMalloc(&d_z,numSamples*sizeof(float));
  hipMalloc(&d_tx,numSamples*sizeof(float));
  hipMalloc(&d_ty,numSamples*sizeof(float));
  hipMalloc(&d_tz,numSamples*sizeof(float));
  hipMalloc(&d_temp,numSamples*sizeof(float));
  hipMalloc(&d_mass,numSamples*sizeof(float));
  hipMalloc(&d_newtx,numSamples*sizeof(float));
  hipMalloc(&d_newty,numSamples*sizeof(float));
  hipMalloc(&d_newtz,numSamples*sizeof(float));
  hipMalloc(&d_newtemp,numSamples*sizeof(float));
  hipMalloc(&d_newmass,numSamples*sizeof(float));
  printf("Filling...\n");
  //Fill values:
  int xi=0;
  int yi=0;
  int zi=0;
  int ai=0;
  for (double alt=LAYER_HEIGHT;alt<=ALTITUDE;alt+=LAYER_HEIGHT) {
    xi=0;
    for (double xc=-1;xc<=1;xc+=4/RES) {
    yi=0;
      for (double yc=-1;yc<=1;yc+=4/RES) {
        zi=0;
        for (double zc=-1;zc<=1;zc+=4/RES) {
          if (xc==-1||xc==1||yc==-1||yc==1||zc==-1||zc==1) {
            //Cube coordinates projected:
            double altAdjust=alt/sqrt(pow(xc,2)+pow(yc,2.0)+pow(zc,2.0));
            printf("here, index %i, xi %i, yi %i, zi %i, and ai %i\n",xi*RES*RES*RES+yi*RES*RES+zi*RES+ai,xi,yi,zi,ai);
            x[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=xc*altAdjust;
            y[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=yc*altAdjust;
            z[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=zc*altAdjust;
            //Trajectories:
            srand ( time ( NULL));
            printf("or here, not that it's any different\n");
            tx[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTrajectory-leastAllowedTrajectory)+leastAllowedTrajectory;
            srand ( time ( NULL)+1);
            ty[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTrajectory-leastAllowedTrajectory)+leastAllowedTrajectory;
            srand ( time ( NULL)+2);
            tz[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTrajectory-leastAllowedTrajectory)+leastAllowedTrajectory;
            //Temp, mass:
            srand ( time ( NULL)+3);
            mass[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedMass-leastAllowedMass)+leastAllowedMass;//Too lazy to change variable names despite grammatical innacuracy.
            srand ( time ( NULL)+4);//I add 1,2,3, and 4 to the seed in case your cpu is a blazing fast juggernaut.
            temp[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTemp-leastAllowedTemp)+leastAllowedTemp;
          }
          zi++;
        }
        yi++;
      }
      xi++;
    }
    ai++;
  }
  printf("Generated, copying to GPU...\n");
  //Copy Values:
  hipMemcpy(d_x,x,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_y,y,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_z,z,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_tx,tx,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_ty,ty,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_tz,tz,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newtx,newtx,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newty,newty,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newtz,newtz,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_temp,temp,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_mass,mass,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newtemp,newtemp,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newmass,newmass,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  printf("Copied. Starting kernel/server loop. Connect to port 2084 in you renderer.\n");
  //"Start server thread." Not anymore, I learned Kernels cannot write to host memory and the cpu cannot pull from kernels while they are running. So I guess we do it in a loop, which is slower but at least physically possible.
  //"pthread_join(thread_id,NULL); // Wait for server (test, don't actually do)." Also irrelevant.
  //Kernels:
  //More irrelevant comments for reference:
  // cudaDeviceProp deviceProp;
  // cudaGetDeviceProperties(&deviceProp,0);//The zero is for GPU zero, as I assume you have not bought extra GPUs for this task.
  // int rate=deviceProp.clockRate;
  //Dumping/Plagiarizing some server code:
  int sockfd;
  char buffer[MAXLINE];
  char *hello = "Hello from server";
  struct sockaddr_in servaddr, cliaddr;

  // Creating socket file descriptor
  if ( (sockfd = socket(AF_INET, SOCK_DGRAM, 0)) < 0 ) {
      perror("socket creation failed");
      exit(EXIT_FAILURE);
  }
  printf("wORkiNg...\n");
  memset(&servaddr, 0, sizeof(servaddr));
  memset(&cliaddr, 0, sizeof(cliaddr));

  // Filling server information
  servaddr.sin_family    = AF_INET; // IPv4
  servaddr.sin_addr.s_addr = INADDR_ANY;
  servaddr.sin_port = htons(PORT);
  printf("WorKInG...\n");
  // Bind the socket with the server address
  if ( bind(sockfd, (const struct sockaddr *)&servaddr,
          sizeof(servaddr)) < 0 )
  {
      perror("bind failed");
      exit(EXIT_FAILURE);
  }
  printf("Beginning Loop\n");
  int len;
  while(true) {
    //Use numSamples for core count, one thread each. In the future we'll want to get multiple threads for when we run out of cores.
    osmate<<<numSamples,1>>>(TEMP_RATE,MASS_RATE,TEMP_PUSH,REPULSION_RATE,numSamples,ALTITUDE,GRAV_CONST,d_x,d_y,d_z,d_tx,d_ty,d_tz,d_newtx,d_newty,d_newtz,d_mass,d_temp,d_newmass,d_newtemp);
    hipMemcpy(temp,d_newtemp,sizeof(double)*numSamples,hipMemcpyDeviceToHost);
    hipMemcpy(mass,d_newmass,sizeof(double)*numSamples,hipMemcpyDeviceToHost);
    hipMemcpy(d_tx,d_newtx,sizeof(double)*numSamples,hipMemcpyDeviceToDevice);
    hipMemcpy(d_ty,d_newty,sizeof(double)*numSamples,hipMemcpyDeviceToDevice);
    hipMemcpy(d_tz,d_newtz,sizeof(double)*numSamples,hipMemcpyDeviceToDevice);
    //Send temp,mass
    sendto(sockfd, (const char *)("1"), 1, MSG_CONFIRM, (const struct sockaddr *) &cliaddr,len);
    for (int i=0;i<numSamples;i++) {
      if (temp[i]!=0) {
        printf("%f\n",temp[i]);
      }
      if (mass[i]!=0) {
        printf("%f\n",mass[i]);
      }
    }
    sendto(sockfd, temp, numSamples, MSG_CONFIRM, (const struct sockaddr *) &cliaddr,len);
    sendto(sockfd, (const char *)("2"), 1, MSG_CONFIRM, (const struct sockaddr *) &cliaddr,len);
    sendto(sockfd, mass, numSamples, MSG_CONFIRM, (const struct sockaddr *) &cliaddr,len);
    //Update so we can go again
    hipMemcpy(d_temp,temp,sizeof(double)*numSamples,hipMemcpyHostToDevice);
    hipMemcpy(d_mass,mass,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  }
}
