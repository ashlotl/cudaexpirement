
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "math.h"
#include <pthread.h>
#include <unistd.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#define PORT    2084
#define MAXLINE 1024
/*
I will provide a word about tx,ty, and tz.
tx does not, in fact move the actual point (points are static), but defines how much mass will be donated.
After this donation, a density-based heat donation is calculated, but regardless of these trajectory values.
*/
__global__
void osmate(double tempRate, double massRate, double tempPush, double repulsionRate, int numPoints, double radius, double gConst, double * x, double * y, double * z, double * tx, double * ty, double * tz, double * newtx, double * newty, double * newtz, double * mass, double * temp, double * newmass, double * newtemp) {
  //The only part that really matters if I get the rest to work:
  int i=blockIdx.x + blockDim.x + threadIdx.x;
  for (int it=0;it<numPoints;it++) {//Iterate through all points
    // printf("%f",tx[it]);
    double dist=sqrt((x[it]-x[i])*(x[it]-x[i])+(y[it]-y[i])*(y[it]-y[i])+(z[it]-z[i])*(z[it]-z[i]));//This could be pre-calculated (these points are static). Whether the calculation happens should be a matter of how many points there are -- if there are too many points we will run out of memory, but if there are not as many it is a good idea. I'm leaving it for another day.
    if (dist!=0) {
      //Congratulations, you are not looking at the same point.
      double toChange=gConst*mass[i]*mass[it]/dist;
      newtx[i]=tx[i]+(toChange+tempPush*(temp[i]-temp[it])+repulsionRate*(mass[i]-mass[it]))*(x[it]-x[i]);
      newty[i]=ty[i]+(toChange+tempPush*(temp[i]-temp[it])+repulsionRate*(mass[i]-mass[it]))*(y[it]-y[i]);
      newtz[i]=tz[i]+(toChange+tempPush*(temp[i]-temp[it])+repulsionRate*(mass[i]-mass[it]))*(z[it]-z[i]);
    }
  }
  for (int it=0;it<numPoints;it++) {
    //TODO make negative mass impossible.
    double massDisagreement=sqrt((x[it]-x[i])*(x[it]-x[i])/tx[i]+(y[it]-y[i])*(y[it]-y[i])/ty[i]+(z[it]-z[i])*(z[it]-z[i])/tz[i]);//TODO cases for rare 0 trajectory scenarios that would knacker it all.
    newmass[it]=mass[it]+massRate/massDisagreement;//Hell if I know whether this is right, but it seems it should intuitively work. Also grants some cohesion. Note we don't subtract from the subject sample because the mass comes from behind it.
    double dist=sqrt((x[it]-x[i])*(x[it]-x[i])+(y[it]-y[i])*(y[it]-y[i])+(z[it]-z[i])*(z[it]-z[i]));//There are a lot of redundant calculations here...
    double tempEx=tempRate*(temp[i]-temp[it])/dist;
    newtemp[it]=temp[it]+tempEx;
    newtemp[i]=temp[i]-tempEx;//Unless we want to go supernova. We should also have a mechanic where a mass increase "creates" heat, a decrease sucks it up, there is an external source of heat and heat can radiate away... Etc.
    // printf("%f",tx[i]);
  }
}
//To tweak
double TEMP_RATE=0.1;
double MASS_RATE=0.4;
double TEMP_PUSH=0.03;
double REPULSION_RATE=1.0;
double SAMPLE_RADIUS=.1;
double GRAV_CONST=.0667;//Definitely not.

int ALTITUDE=5;//Radius of planet.
double LAYER_HEIGHT=.5;
int RES = 3;

//In generation, values may exceed maximums and minimums during simulation.
double leastAllowedMass=.001;
double mostAllowedMass=5;//We'll randomize between the two.

double leastAllowedTemp=.001;//Kelvin?
double mostAllowedTemp=50000;//A lot, hopefully not too much. Should check Planck heat.

double leastAllowedTrajectory=.0001;//Along axis, I'm not a madman
double mostAllowedTrajectory=2;
void delay(int number_of_millis)
{
    // Converting time into milli_seconds
    // int milli_seconds = 1000 * number_of_seconds;

    // Storing start time
    clock_t start_time = clock();

    // looping till required time is not acheived
    while (clock() < start_time + number_of_millis)
        ;
}
void serialize_float(char *buffer, float value)
{
    unsigned int ivalue = *((unsigned int*)&value); // warning assumes 32-bit "unsigned int"
    buffer[0] = ivalue >> 24;
    buffer[1] = ivalue >> 16;
    buffer[2] = ivalue >> 8;
    buffer[3] = ivalue;
}
char* doublearraytochar(double * vals, int len) {
  char * result=(char *)malloc(len*sizeof(char)*4);
  for (int i=0;i<len;i++) {
    char * tempArr=(char *)malloc(sizeof(char)*4);
    serialize_float(tempArr,vals[i]);
    for (int i2=0;i2<4;i2++) {
      result[i*4+i2]=tempArr[i2];
    }
  }
  return result;
}
char * jsonArray(double * vals, int len, int bufPerNum, char * result, char * output) {
  result[0]='[';
  for (int i=0;i<len;i++) {
    if (i!=len-1) {
      snprintf(output,bufPerNum*2,"%f,",vals[i]);
    } else {
      snprintf(output,bufPerNum*2,"%f]",vals[i]);
    }
    strcat(result,output);
    // for (int i2=0;i2<(bufPerNum+1)*sizeof(char);i2++) {
    //   result[i*bufPerNum+i2+1]=output[i2];
    // }
  }
  return result;
}
int main() {
  srand ( time ( NULL));
  printf("Hello, you are hopefully simulating tectonics. If not, get out of here.\n");
  //Generate points...
  //The Declaration of Variable Allocation:
  int numSamples=0;
  for (double alt=LAYER_HEIGHT;alt<ALTITUDE;alt+=LAYER_HEIGHT) {
    //You may notice we project a cube into a sphere. Primarily I'm too lazy to generate something like an icosahedron, secondarily it would not have fine resolution (only subdivisions) if we used an icosahedron, tertiarily this allows us to check for seams, which are screaming alarms something is wrong, and fourth uv spheres get ridiculously fine detail only at the poles, which is a waste.
    numSamples+=(int)(pow(RES,3)-pow(RES-2,3));
  }
  printf("Allocating for %i samples (pass to your renderer your RES, %i, and your ALTITUDE, %i, like so: python display.py <RES> <ALTITUDE>)...\n", numSamples, RES,ALTITUDE);
  double * x = (double*)malloc(numSamples*sizeof(double));
  double * y = (double*)malloc(numSamples*sizeof(double));
  double * z = (double*)malloc(numSamples*sizeof(double));
  double * tx = (double*)malloc(numSamples*sizeof(double));
  double * ty = (double*)malloc(numSamples*sizeof(double));
  double * tz = (double*)malloc(numSamples*sizeof(double));
  double * temp = (double*)malloc(numSamples*sizeof(double));
  double * mass = (double*)malloc(numSamples*sizeof(double));
  double * newtx = (double*)malloc(numSamples*sizeof(double));
  double * newty = (double*)malloc(numSamples*sizeof(double));
  double * newtz = (double*)malloc(numSamples*sizeof(double));
  double * newtemp = (double*)malloc(numSamples*sizeof(double));
  double * newmass = (double*)malloc(numSamples*sizeof(double));
  //Here we have device copies:
  double * d_x;
  double * d_y;
  double * d_z;
  double * d_tx;
  double * d_ty;
  double * d_tz;
  double * d_temp;
  double * d_mass;
  double * d_newtx;
  double * d_newty;
  double * d_newtz;
  double * d_newtemp;
  double * d_newmass;
  hipMalloc(&d_x,numSamples*sizeof(float));
  hipMalloc(&d_y,numSamples*sizeof(float));
  hipMalloc(&d_z,numSamples*sizeof(float));
  hipMalloc(&d_tx,numSamples*sizeof(float));
  hipMalloc(&d_ty,numSamples*sizeof(float));
  hipMalloc(&d_tz,numSamples*sizeof(float));
  hipMalloc(&d_temp,numSamples*sizeof(float));
  hipMalloc(&d_mass,numSamples*sizeof(float));
  hipMalloc(&d_newtx,numSamples*sizeof(float));
  hipMalloc(&d_newty,numSamples*sizeof(float));
  hipMalloc(&d_newtz,numSamples*sizeof(float));
  hipMalloc(&d_newtemp,numSamples*sizeof(float));
  hipMalloc(&d_newmass,numSamples*sizeof(float));
  printf("Filling...\n");
  //Fill values:
  int xi=0;
  int yi=0;
  int zi=0;
  int ai=0;
  for (double alt=LAYER_HEIGHT;alt<ALTITUDE;alt+=LAYER_HEIGHT) {
    xi=0;
    for (double xc=-1;xc<=1;xc+=2/(RES-1)) {
    yi=0;
      for (double yc=-1;yc<=1;yc+=2/(RES-1)) {
        zi=0;
        for (double zc=-1;zc<=1;zc+=2/(RES-1)) {
          if (xc==-1||xc==1||yc==-1||yc==1||zc==-1||zc==1) {
            //Cube coordinates projected:
            double altAdjust=alt/sqrt(pow(xc,2)+pow(yc,2.0)+pow(zc,2.0));
            x[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=xc*altAdjust;
            y[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=yc*altAdjust;
            z[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=zc*altAdjust;
            //Trajectories:
            tx[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTrajectory-leastAllowedTrajectory)+leastAllowedTrajectory;
            ty[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTrajectory-leastAllowedTrajectory)+leastAllowedTrajectory;
            tz[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTrajectory-leastAllowedTrajectory)+leastAllowedTrajectory;
            //Temp, mass:
            mass[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedMass-leastAllowedMass)+leastAllowedMass;//Too lazy to change variable names despite grammatical innacuracy.
            temp[xi*RES*RES*RES+yi*RES*RES+zi*RES+ai]=((double)rand()/RAND_MAX)*(mostAllowedTemp-leastAllowedTemp)+leastAllowedTemp;
          }
          zi++;
        }
        yi++;
      }
      xi++;
    }
    ai++;
  }
  printf("Generated, copying to GPU...\n");
  //Copy Values:
  hipMemcpy(d_x,x,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_y,y,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_z,z,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_tx,tx,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_ty,ty,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_tz,tz,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newtx,newtx,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newty,newty,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newtz,newtz,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_temp,temp,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_mass,mass,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newtemp,newtemp,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  hipMemcpy(d_newmass,newmass,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  printf("Copied. Starting kernel/server loop. Connect to port 2084 in you renderer.\n");
  //"Start server thread." Not anymore, I learned Kernels cannot write to host memory and the cpu cannot pull from kernels while they are running. So I guess we do it in a loop, which is slower but at least physically possible.
  //"pthread_join(thread_id,NULL); // Wait for server (test, don't actually do)." Also irrelevant.
  //Kernels:
  //More irrelevant comments for reference:
  // cudaDeviceProp deviceProp;
  // cudaGetDeviceProperties(&deviceProp,0);//The zero is for GPU zero, as I assume you have not bought extra GPUs for this task.
  // int rate=deviceProp.clockRate;
  //Dumping/Plagiarizing some server code:
  int sockfd;
  char buffer[numSamples*32];
  char *hello = "Hello from server";
  struct sockaddr_in servaddr, cliaddr;

  // Creating socket file descriptor
  if ( (sockfd = socket(AF_INET, SOCK_DGRAM, 0)) < 0 ) {
      perror("socket creation failed");
      exit(EXIT_FAILURE);
  }
  printf("wORkiNg...\n");
  memset(&servaddr, 0, sizeof(servaddr));
  memset(&cliaddr, 0, sizeof(cliaddr));

  // Filling server information
  servaddr.sin_family    = AF_INET; // IPv4
  // inet_pton(AF_INET, GetIP().c_str(), &(servaddr.sin_addr));
  servaddr.sin_addr.s_addr = INADDR_ANY;
  servaddr.sin_port = htons(PORT);
  sockfd=socket(AF_INET,SOCK_DGRAM,0);
  if (sockfd<0) {
    perror("Error on socket()");
  }
  int optval = 1;
  setsockopt(sockfd,SOL_SOCKET,SO_REUSEADDR,(const void *)&optval,sizeof(int));

  printf("WorKInG...\n");
  // Bind the socket with the server address
  if ( bind(sockfd, (const struct sockaddr *)&servaddr,
          sizeof(servaddr)) < 0 )
  {
      perror("bind failed");
      exit(EXIT_FAILURE);
  }
  printf("Beginning Loop\n");
  int len,n;
  n = recvfrom(sockfd, (char *)buffer, MAXLINE,
                MSG_WAITALL, ( struct sockaddr *) &cliaddr,
                (socklen_t *)&len);
  buffer[n]='\0';
  printf("%s,%i",buffer,n);
  char * one=(char*)calloc(numSamples*32,sizeof(char));
  one[0]='1';
  char * two=(char*)calloc(numSamples*32,sizeof(char));
  two[0]='2';
  char * result=(char*)calloc((numSamples*33+1),sizeof(char));
  char * output=(char *)calloc((numSamples*32+1),sizeof(char));
  while(n>0) {
    //Use numSamples for core count, one thread each. In the future we'll want to get multiple threads for when we run out of cores.
    osmate<<<numSamples,1>>>(TEMP_RATE,MASS_RATE,TEMP_PUSH,REPULSION_RATE,numSamples,ALTITUDE,GRAV_CONST,d_x,d_y,d_z,d_tx,d_ty,d_tz,d_newtx,d_newty,d_newtz,d_mass,d_temp,d_newmass,d_newtemp);
    hipDeviceSynchronize();
    // printf("Synchronized.\n");
    hipMemcpy(temp,d_newtemp,sizeof(double)*numSamples,hipMemcpyDeviceToHost);
    hipMemcpy(mass,d_newmass,sizeof(double)*numSamples,hipMemcpyDeviceToHost);
    hipMemcpy(d_newtx,d_tx,sizeof(double)*numSamples,hipMemcpyDeviceToDevice);
    hipMemcpy(d_newty,d_ty,sizeof(double)*numSamples,hipMemcpyDeviceToDevice);
    hipMemcpy(d_newtz,d_tz,sizeof(double)*numSamples,hipMemcpyDeviceToDevice);
    //Send temp,mass
    // delay(10000);
    sendto(sockfd, one, numSamples*32, MSG_DONTWAIT, (const struct sockaddr *) &cliaddr,len);
    // delay(10000);
    // printf(jsonArray(temp,numSamples,numSamples*32));
    // printf("\n");
    jsonArray(temp,numSamples,numSamples*32,result,output);
    sendto(sockfd, result, numSamples*32, MSG_DONTWAIT, (const struct sockaddr *) &cliaddr,len);
    // delay(10000);
    sendto(sockfd, two, numSamples*32, MSG_DONTWAIT, (const struct sockaddr *) &cliaddr,len);
    // delay(10000);
    jsonArray(mass,numSamples,numSamples*32,result,output);
    sendto(sockfd, result, numSamples*32, MSG_DONTWAIT, (const struct sockaddr *) &cliaddr,len);
    //Update so we can go again
    hipMemcpy(d_temp,temp,sizeof(double)*numSamples,hipMemcpyHostToDevice);
    hipMemcpy(d_mass,mass,sizeof(double)*numSamples,hipMemcpyHostToDevice);
  }
}
